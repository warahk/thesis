#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>

// include ispc's timing.h for timing
#include "../ispc_spheres/timing.h"
// include rand_sphere for generating spheres
#include "../common/rand_sphere.h"
#define MAX_RAY_DEPTH 5


struct Sphere {
    float center[3];
    float radius, radius2;
    float transparency, reflection;
    float emissionColor[3], surfaceColor[3];
};


__device__ void Normalize(float3 &v1) {
    float nor2 = v1.x * v1.x + v1.y * v1.y + v1.z * v1.z;
    if (nor2 > 0) {
        float invNor = 1 / sqrt(nor2);
        v1.x *= invNor, v1.y *= invNor, v1.z *= invNor;
    }
}

__device__ float3 Cross(const float3 v1, const float3 v2) {
    float v1x = v1.x, v1y = v1.y, v1z = v1.z;
    float v2x = v2.x, v2y = v2.y, v2z = v2.z;
    float3 ret;
    ret.x = (v1y * v2z) - (v1z * v2y);
    ret.y = (v1z * v2x) - (v1x * v2z);
    ret.z = (v1x * v2y) - (v1y * v2x);
    return ret;
}

__device__ float Dot(const float3 a, const float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float mix(const float &a, const float &b, const float &mix) {
    return b * mix + a * (1 - mix);
}

__device__ float3 operator+(const float3 &a, const float3 &b) {
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
	return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float3 operator*(const float3 &a, const float3 &b) {
	return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

__device__ float3 operator*(const float3 &a, const float &b) {
	return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ bool SphereIntersect(const Sphere &sphere,
                            const float3 &rayorig,
                            const float3 &raydir,
                            float &t0, float &t1)  {
    float3 center = make_float3(sphere.center[0], sphere.center[1], sphere.center[2]);
    float3 l = center - rayorig;
    float tca = Dot(l, raydir);
    if (tca < 0) return false;
    float d2 = Dot(l, l) - tca * tca;
    if (d2 > sphere.radius2) return false;
    float thc = sqrt(sphere.radius2 - d2);
    t0 = tca - thc;
    t1 = tca + thc;
    return true;
}


 __device__ float3 trace(const float3 &rayorig, const float3 &raydir, const Sphere *spheres, const int depth, const int size) {
    float tnear = INFINITY;
    Sphere sphere;
    bool intersect_found = false;
    for (int i = 0; i < size; ++i) {
        float t0 = INFINITY, t1 = INFINITY;
        if (SphereIntersect(spheres[i], rayorig, raydir, t0, t1)) {
            if (t0 < 0) t0 = t1;
            if (t0 < tnear) {
                tnear = t0;
                sphere = spheres[i];
                intersect_found = true;
            }
        }
    }
    // if there's no intersection return black or background color
    if (!intersect_found) {
        float3 ret = make_float3(2, 2, 2);
        return ret;
    }
    float3 surfaceColor = make_float3(0, 0, 0);
    float3 phit = rayorig + raydir * tnear; // point of intersection
    float3 center = make_float3(sphere.center[0], sphere.center[1], sphere.center[2]);
    float3 nhit = phit - center; // normal at the insection point

    Normalize(nhit); // normalize the direction
    // If the normal and the view direction are not opposite to each other
    // reverse the normal direction. That also means we are inside the sphere so set
    // the inside bool to true. Finally reverse the sign of IdotN which we want
    // positive.
    float bias = 1e-4;
    bool inside = false;

    if (Dot(raydir, nhit) > 0) nhit = nhit * -1.0, inside = true;
    if ((sphere.transparency > 0 || sphere.reflection > 0) && depth < MAX_RAY_DEPTH) {
        float facingratio = -(Dot(raydir, nhit));
        // change the mix value to the tweak effect
        const float fresneleffect = mix(powf(1 - facingratio, 3), 1.0, 0.1);
        // compute reflection direction (no need to normalize because all vectors
        // are already normalized)
        float3 refldir = raydir - nhit * 2 * Dot(raydir, nhit);
        Normalize(refldir);
        float3 reflection = trace(phit + nhit * bias, refldir, spheres, depth + 1, size);
        float3 refraction = make_float3(0,0,0);
        // if sphere is also transparent, compute refraction ray
        if (sphere.transparency) {
            float ior = 1.1;
            float eta;
            if (inside) eta = ior;
            else eta = 1 / ior; 
            float cosi = -(Dot(nhit, raydir));
            float k = 1 - eta * eta * (1 - cosi * cosi);
            float3 refrdir = raydir * eta + nhit * (eta * cosi - sqrtf(k));
            Normalize(refrdir);
            refraction = trace(phit - nhit * bias, refrdir, spheres, depth + 1, size);
        }
        // the result is a mix of reflection and refraction 
        float3 sphereSurfaceColor = make_float3(sphere.surfaceColor[0],
                                     			sphere.surfaceColor[1],
                                     			sphere.surfaceColor[2]);
        surfaceColor = (
            reflection * fresneleffect +
            refraction * (1 - fresneleffect) * sphere.transparency) * sphereSurfaceColor;
    }
    else {
        // it's a diffuse object, no need to raytace any further
        for (int i = 0; i < size; ++i) {
            if (spheres[i].emissionColor[0] > 0) {
                // this is a light
                float3 transmission = make_float3(1,1,1);
                float3 centeri = make_float3(spheres[i].center[0],
                                  			 spheres[i].center[1],
                                  		     spheres[i].center[2]);
                float3 lightDirection = centeri - phit;
                Normalize(lightDirection);
                for (int j = 0; j < size; ++j) {
                    if (i != j) {
                        float t0, t1;
                        if (SphereIntersect(spheres[j], phit + nhit * bias,
                                                    lightDirection, t0, t1)) {
                            transmission = make_float3(0,0,0);
                            break;
                        }
                    }
                }
                float3 sphereSurfaceColor = make_float3(
											 sphere.surfaceColor[0],
                                             sphere.surfaceColor[1],
                                             sphere.surfaceColor[2]);

                float3 sphereEmissionColor = make_float3(
                                             spheres[i].emissionColor[0],
                                             spheres[i].emissionColor[1],
                                             spheres[i].emissionColor[2]);
                surfaceColor = surfaceColor + sphereSurfaceColor * transmission * max(0., Dot(nhit, lightDirection)) * sphereEmissionColor;
            }
        }
    }
    float3 sphereEmissionColor = make_float3(
                             sphere.emissionColor[0],
                             sphere.emissionColor[1],
                             sphere.emissionColor[2]);
    return surfaceColor + sphereEmissionColor;
}


__global__ void render(float *gpu_pixels_r,
					   float *gpu_pixels_g,
				       float *gpu_pixels_b,
					   const int width,
					   const int height,
					   Sphere * spheres,
					   const int sphere_count,
					   const float invWidth,
					   const float invHeight,
					   const float fov,
					   const float aspectratio,
					   const float angle){
    int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(pos_x >= width || pos_y >= height)
        return;        
    int pixel_index = pos_x + pos_y * width;
	float xx = (2 * ((pos_x + 0.5) * invWidth) - 1) * angle * aspectratio;
	float yy = (1 - 2 * ((pos_y + 0.5) * invHeight)) * angle;
	float3 raydir = make_float3(xx, yy, -1);
	Normalize(raydir);
	float3 ray_orig = make_float3(0, 0, 0);
	float3 pixel_color = trace(ray_orig, raydir, spheres, 0, sphere_count);
	gpu_pixels_r[pixel_index] = pixel_color.x;
	gpu_pixels_g[pixel_index] = pixel_color.y;
	gpu_pixels_b[pixel_index] = pixel_color.z;
}


int main(int argc, char **argv) {
    // parse command line args
    if (argc != 4 && argc != 1) {
        std::cerr << "Usage: " << argv[0] << " <width> <height> <sphere_count>" << std::endl;
        return EXIT_FAILURE;
    }

	Sphere *cpuSpheres;
	Sphere *gpuSpheres;
	int sphereSize = sizeof(Sphere);
    int width, height, numSpheres, numBytes;
		
	// Define spheres
    if (argc == 1) { 
	    // Allocate memory for Sphere arrays
        numSpheres = 6;
        numBytes = numSpheres * sphereSize;
        cpuSpheres = (Sphere*)malloc(numBytes);
        hipMalloc((void**)&gpuSpheres, numBytes);

        cpuSpheres[0].center[0] = 0.0;
        cpuSpheres[0].center[1] = -10004;
        cpuSpheres[0].center[2] = -20;
        cpuSpheres[0].radius = 10000;
        cpuSpheres[0].radius2 = 10000 * 10000;
        cpuSpheres[0].surfaceColor[0] = 0.20;
        cpuSpheres[0].surfaceColor[1] = 0.20;
        cpuSpheres[0].surfaceColor[2] = 0.20;
        cpuSpheres[0].reflection = 0.0;
        cpuSpheres[0].transparency = 0.0;
        cpuSpheres[0].emissionColor[0] = 0.0;
        cpuSpheres[0].emissionColor[1] = 0.0;
        cpuSpheres[0].emissionColor[2] = 0.0;

        cpuSpheres[1].center[0] = 0.0;
        cpuSpheres[1].center[1] = 0;
        cpuSpheres[1].center[2] = -20;
        cpuSpheres[1].radius = 4;
        cpuSpheres[1].radius2 = 16;
        cpuSpheres[1].surfaceColor[0] = 1;
        cpuSpheres[1].surfaceColor[1] = 0.32;
        cpuSpheres[1].surfaceColor[2] = 0.36;
        cpuSpheres[1].reflection = 1;
        cpuSpheres[1].transparency = 0.5;
        cpuSpheres[1].emissionColor[0] = 0.0;
        cpuSpheres[1].emissionColor[1] = 0.0;
        cpuSpheres[1].emissionColor[2] = 0.0;
        
        cpuSpheres[2].center[0] = 5;
        cpuSpheres[2].center[1] = -1;
        cpuSpheres[2].center[2] = -15;
        cpuSpheres[2].radius = 2;
        cpuSpheres[2].radius2 = 4;
        cpuSpheres[2].surfaceColor[0] = .90;
        cpuSpheres[2].surfaceColor[1] = 0.76;
        cpuSpheres[2].surfaceColor[2] = 0.46;
        cpuSpheres[2].reflection = 1;
        cpuSpheres[2].transparency = 0;
        cpuSpheres[2].emissionColor[0] = 0.0;
        cpuSpheres[2].emissionColor[1] = 0.0;
        cpuSpheres[2].emissionColor[2] = 0.0;
        
        cpuSpheres[3].center[0] = 5;
        cpuSpheres[3].center[1] = 0;
        cpuSpheres[3].center[2] = -25;
        cpuSpheres[3].radius = 3;
        cpuSpheres[3].radius2 = 9;
        cpuSpheres[3].surfaceColor[0] = 0.65;
        cpuSpheres[3].surfaceColor[1] = 0.77;
        cpuSpheres[3].surfaceColor[2] = 0.97;
        cpuSpheres[3].reflection = 1;
        cpuSpheres[3].transparency = 0;
        cpuSpheres[3].emissionColor[0] = 0.0;
        cpuSpheres[3].emissionColor[1] = 0.0;
        cpuSpheres[3].emissionColor[2] = 0.0;

        cpuSpheres[4].center[0] = -5.5;
        cpuSpheres[4].center[1] = 0;
        cpuSpheres[4].center[2] = -15;
        cpuSpheres[4].radius = 3;
        cpuSpheres[4].radius2 = 9;
        cpuSpheres[4].surfaceColor[0] = 0.90;
        cpuSpheres[4].surfaceColor[1] = 0.90;
        cpuSpheres[4].surfaceColor[2] = 0.90;
        cpuSpheres[4].reflection = 1;
        cpuSpheres[4].transparency = 0;
        cpuSpheres[4].emissionColor[0] = 0.0;
        cpuSpheres[4].emissionColor[1] = 0.0;
        cpuSpheres[4].emissionColor[2] = 0.0;
    }
    else {
        // Read in argv
        height = std::atoi(argv[1]);
        width = std::atoi(argv[2]);
        numSpheres = std::atoi(argv[3]);
	    // Allocate memory for Sphere arrays
        numBytes = numSpheres * sphereSize;
        cpuSpheres = (Sphere*)malloc(numBytes);
        hipMalloc((void**)&gpuSpheres, numBytes);
        // Generate random spheres within frustum
        srand(time(NULL));
        float x,y,z;
        int count = 0;
	    while (count < numSpheres - 1) {
            rand_sphere(x, y, z, height, width, 2);
            cpuSpheres[count].center[0] = x;
            cpuSpheres[count].center[1] = y;
            cpuSpheres[count].center[2] = z;
            cpuSpheres[count].radius = 2;
            cpuSpheres[count].radius2 = 4;
            cpuSpheres[count].surfaceColor[0] = random_float(0,1);
            cpuSpheres[count].surfaceColor[1] = random_float(0,1);
            cpuSpheres[count].surfaceColor[2] = random_float(0,1);
            cpuSpheres[count].reflection = 0.5;
            cpuSpheres[count].transparency = 0.5;
            cpuSpheres[count].emissionColor[0] = 0.0;
            cpuSpheres[count].emissionColor[1] = 0.0;
            cpuSpheres[count].emissionColor[2] = 0.0;
            count++;
        }
    }   
    

	// light
    int last = numSpheres - 1;
	cpuSpheres[last].center[0] = 0.0;
	cpuSpheres[last].center[1] = 20;
	cpuSpheres[last].center[2] = -30;
	cpuSpheres[last].radius = 3;
	cpuSpheres[last].radius2 = 9;
	cpuSpheres[last].surfaceColor[0] = 0.0;
	cpuSpheres[last].surfaceColor[1] = 0.0;
	cpuSpheres[last].surfaceColor[2] = 0.0;
	cpuSpheres[last].reflection = 0;
	cpuSpheres[last].transparency = 0;
	cpuSpheres[last].emissionColor[0] = 3.0;
	cpuSpheres[last].emissionColor[1] = 3.0;
	cpuSpheres[last].emissionColor[2] = 3.0;

	
	// Allocate memory for pixel arrays
	float *pixel_out_r = new float[width * height]; 
	float *pixel_out_g = new float[width * height]; 
	float *pixel_out_b = new float[width * height]; 
	float *gpu_pixel_r;	
	float *gpu_pixel_g;
	float *gpu_pixel_b;
	hipMalloc((void**)&gpu_pixel_r, sizeof(float)*width*height); 
	hipMalloc((void**)&gpu_pixel_g, sizeof(float)*width*height); 
	hipMalloc((void**)&gpu_pixel_b, sizeof(float)*width*height); 

	// Derive display variables
	float invWidth = 1 / (float)width;
	float invHeight = 1 / (float)height;
    float fov = 30;
    float aspectratio = width / (float)height;
    float angle = tan(M_PI * 0.5 * fov / 180.);
	
	reset_and_start_timer();
	// Copy cpuSpheres to gpuSpheres
	hipMemcpy(gpuSpheres, cpuSpheres, numBytes, hipMemcpyHostToDevice); 
	 
	const dim3 blockSize(24,24,1);
	const dim3 gridSize(width/24+1, height/24+1, 1);
	render<<<gridSize,blockSize>>>(gpu_pixel_r, gpu_pixel_g, gpu_pixel_b, width, height, gpuSpheres, numSpheres, invWidth, invHeight, fov, aspectratio, angle);

	// Copy arrays back to cpu
	hipMemcpy(pixel_out_r, gpu_pixel_r, sizeof(float)*width*height, hipMemcpyDeviceToHost);
	hipMemcpy(pixel_out_g, gpu_pixel_g, sizeof(float)*width*height, hipMemcpyDeviceToHost);
	hipMemcpy(pixel_out_b, gpu_pixel_b, sizeof(float)*width*height, hipMemcpyDeviceToHost);
	// Stop timer
	double dt = get_elapsed_mcycles();
	printf("@time of CUDA run:\t\t\t[%.3f] million cycles\n", dt);

	
    // Save result to a PPM image (keep these flags if you compile under Windows)
    std::ofstream ofs("./CUDA_spheres.ppm", std::ios::out | std::ios::binary);
    ofs << "P6\n" << width << " " << height << "\n255\n";
    for (unsigned i = 0; i < width * height; ++i) {
        ofs << (unsigned char)(std::min(float(1), pixel_out_r[i]) * 255) <<
               (unsigned char)(std::min(float(1), pixel_out_g[i]) * 255) <<
               (unsigned char)(std::min(float(1), pixel_out_b[i]) * 255);
    }   
    ofs.close();

	// Deallocate memory
    delete [] pixel_out_r;
    delete [] pixel_out_g;
    delete [] pixel_out_b;
	free(cpuSpheres);
	hipFree(gpuSpheres);
	hipFree(gpu_pixel_r);
	hipFree(gpu_pixel_g);
	hipFree(gpu_pixel_b);
	
	return 0;
}

	
	
